
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  // printf("Thread thread ID, %d\n", tid);
    if (tid < n) {
      out[tid] = a[tid] + b[tid];
    }
}

int main(){
    float *a, *b, *out;       // host memory
    float *d_a, *d_b, *d_out; // device memory 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate device memory for a, b & out
    hipMalloc((void**)&d_a,   sizeof(float) * N);
    hipMalloc((void**)&d_b,   sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a,      sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b,      sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    // Transfer data back from device to host memory
    hipMemcpy(out, d_out,  sizeof(float) * N, hipMemcpyDeviceToHost);

    // verify result
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");
    // cudaDeviceSynchronize();

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
