
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;       // host memory
    float *d_a, *d_b, *d_out; // device memory 

    // Allocate memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Allocate device memory for a, b & out
    hipMalloc((void**)&d_a,   sizeof(float) * N);
    hipMalloc((void**)&d_b,   sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a,      sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b,      sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);

    // Transfer data back from device to host memory
    hipMemcpy(out, d_out,  sizeof(float) * N, hipMemcpyDeviceToHost);

    // verify result
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");
    // cudaDeviceSynchronize();

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);
}
